#include "hip/hip_runtime.h"
#ifndef THTENSOR_CUDA_OPS_H
#define THTENSOR_CUDA_OPS_H

// define some additional CUDA operations
extern "C" {
#include "THCudaTensorOps.h"
}
#include "THC/THCApply.cuh"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#define SOFTMAX_THREADS 128

struct TensorDTanOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = 1.- (*in) * (*in);
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = 1.- (*v) * (*v);
	  }
};

struct TensorSigmoidOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = 1./(1.+ exp(- *in));
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = 1./(1.+ exp(- *v));
	  }
};

struct TensorDSigmoidOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (1. - *in) * (*in);
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (1. - *v) * (*v);
	  }
};

struct TensorThresholdOp {
	  TensorThresholdOp(float t, float c, float o) : thresh(t),coeff(c),offset(o) {}

	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (*in) > thresh ? (*in) : coeff * (*in) + offset;
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (*v) > thresh ? (*v) : coeff * (*v) + offset;
	  }
	  
	  const float thresh;
	  const float coeff;
	  const float offset;
};

struct TensorDThresholdOp {
	  TensorDThresholdOp(float t, float c) : thresh(t),coeff(c) {}

	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (*in) > thresh ? 1 : coeff;
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (*v) > thresh ? 1 : coeff;
	  }
	  
	  const float thresh;
	  const float coeff;
};

__global__ void maxpool(float *input, float *output,
                        int input_n, int input_h, int input_w,
                        int kH, int kW, int dH, int dW)
{
	// iterators
	int xx, yy;

	// output size
	const int output_w = (input_w - kW) / dW + 1;
	const int output_h = (input_h - kH) / dH + 1;

	// compute offsets based on thread/block ID
	int o = blockIdx.x;
	int i = o;

	int xx_start = threadIdx.x;
	int xx_end = output_w;
	const int xx_step = blockDim.x;

	int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
	int yy_end = output_h;
	const int yy_step = blockDim.y*gridDim.y;

	// select input/output plane
	output = output + o*output_w*output_h;
	input = input + i*input_w*input_h;

	// For all output pixels...
	for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    	for(xx = xx_start; xx < xx_end; xx+=xx_step) {
			float *ptr_input = input + yy*dH*input_w + xx*dW;
      		float *ptr_output = output + yy*output_w + xx;
      		float max = -FLT_MAX;
      		int kx, ky;
      		for(ky = 0; ky < kH; ky++) {
        		for(kx = 0; kx < kW; kx++) {
          			float val = ptr_input[kx];
          			if (val > max) {
            			max = val;
          			}
        		}
        		ptr_input += input_w; // next input line
      		}
      		// Update output
      		*ptr_output = max;
    	}
  	}
}


// softmax kernel with 128 threads based on cunn Softmax.cu
__global__ void softmax(float *output, float *input, int nframe, int dim)
{
  __shared__ float buffer[ SOFTMAX_THREADS + 1];
  int k = blockIdx.x;
  float *input_k = input + k*dim;
  float *output_k = output + k*dim;

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  // max?
  buffer[threadIdx.x] = -FLT_MAX;
  for (int i=i_start; i<i_end; i+=i_step)
  {
    float z = input_k[i];
    if(buffer[threadIdx.x] < z)
      buffer[threadIdx.x] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float max_k = -FLT_MAX;
    for (int i=0; i<blockDim.x; i++)
    {
      if(max_k < buffer[i])
        max_k = buffer[i];
    }
    buffer[SOFTMAX_THREADS] = max_k;
  }

  __syncthreads();

  // sum?
  float max_k = buffer[SOFTMAX_THREADS];
  buffer[threadIdx.x] = 0;
  for (int i=i_start; i<i_end; i+=i_step) {
    float z = __expf(input_k[i]-max_k);
    buffer[threadIdx.x] += z;
    output_k[i] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float sum_k = 0;
    for (int i=0; i<blockDim.x; i++)
      sum_k += buffer[i];
    buffer[SOFTMAX_THREADS] = sum_k;
  }

  __syncthreads();

  // softmax
  float sum_k = buffer[SOFTMAX_THREADS];
  for (int i=i_start; i<i_end; i+=i_step)
    output_k[i] = output_k[i] / sum_k;
}


extern "C" {
	void THCudaTensor_dtanh(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDTanOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDTanOp());
		}
	}
	
	void THCudaTensor_sigmoid(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorSigmoidOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorSigmoidOp());
		}
	}
	
	void THCudaTensor_dsigmoid(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDSigmoidOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDSigmoidOp());
		}
	}
	
	void THCudaTensor_threshold(THCState *state, THCudaTensor *dest, THCudaTensor* src, float thresh, float coeff, float of){
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorThresholdOp(thresh, coeff, of));
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorThresholdOp(thresh, coeff, of));
		}
	}

	void THCudaTensor_dthreshold(THCState *state, THCudaTensor *dest, THCudaTensor* src, float thresh, float coeff){
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDThresholdOp(thresh, coeff));
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDThresholdOp(thresh, coeff));
		}
	}
	
	int THCudaTensor_argmax(THCState *state, THCudaTensor *t){
		t = THCudaTensor_newContiguous(state, t);
		thrust::device_ptr<float> data(THCudaTensor_data(state, t));

		thrust::device_vector<float>::iterator iter =
			thrust::max_element(data, data + THCudaTensor_nElement(state, t));

		int position = thrust::device_pointer_cast(&(iter[0])) - data;
		THCudaTensor_free(state, t);

		return position;
	}

	int THCudaTensor_argmin(THCState *state, THCudaTensor *t){
		t = THCudaTensor_newContiguous(state, t);
		thrust::device_ptr<float> data(THCudaTensor_data(state, t));

		thrust::device_vector<float>::iterator iter =
			thrust::min_element(data, data + THCudaTensor_nElement(state, t));

		int position = thrust::device_pointer_cast(&(iter[0])) - data;
		THCudaTensor_free(state, t);

		return position;
	}
	
	void THCudaTensor_spatialmaxpool(THCState *state, THCudaTensor *output, THCudaTensor *input,
			int kW, int kH, int dW, int dH){	
		
		long nInputCols = input->size[2];
    	long nInputRows = input->size[1];
    	long nInputPlane = input->size[0];
    	long nOutputCols = (nInputCols - kW) / dW + 1;
    	long nOutputRows = (nInputRows - kH) / dH + 1;

    	input = THCudaTensor_newContiguous(state, input);
    	float* input_data = THCudaTensor_data(state, input);

    	THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
    	float* output_data = THCudaTensor_data(state, output);

    	// cuda blocks & threads:
    	int yblocks = (int)(16L / nInputPlane);
    	yblocks = yblocks < 1 ? 1 : yblocks;
    	dim3 blocks(nInputPlane,yblocks);
    	dim3 threads(32,8);

    	// run maxpool kernel
    	maxpool <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
    	  input_data, output_data,
    	  nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
	
		THCudaTensor_free(state, input);
	}
	
	
	void THCudaTensor_softmax(THCState *state, THCudaTensor *output, THCudaTensor *input){	
    	input = THCudaTensor_newContiguous(state, input);
    	float* input_data = THCudaTensor_data(state, input);
    	float* output_data = THCudaTensor_data(state, output);

    	// cuda blocks & threads:
    	dim3 blocks(1);
    	dim3 threads(SOFTMAX_THREADS);

    	// run softmax kernel
    	softmax<<<blocks,threads,0, THCState_getCurrentStream(state)>>>(
    		output_data, input_data, 1, input->storage->size);
    	
		THCudaTensor_free(state, input);
	}
}
#endif
