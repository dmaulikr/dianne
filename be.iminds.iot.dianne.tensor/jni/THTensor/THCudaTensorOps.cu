#include "hip/hip_runtime.h"
#ifndef THTENSOR_CUDA_OPS_H
#define THTENSOR_CUDA_OPS_H

// define some additional CUDA operations
extern "C" {
#include "THCudaTensorOps.h"
}
#include "THCudaTensorJNI.h"
#include "THC/THCApply.cuh"

struct TensorDTanOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = 1.- (*in) * (*in);
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = 1.- (*v) * (*v);
	  }
};

struct TensorSigmoidOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = 1./(1.+ exp(- *in));
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = 1./(1.+ exp(- *v));
	  }
};

struct TensorDSigmoidOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (1. - *in) * (*in);
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (1. - *v) * (*v);
	  }
};


extern "C" {
	void THCudaTensor_dtanh(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDTanOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDTanOp());
		}
	}
	
	void THCudaTensor_sigmoid(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorSigmoidOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorSigmoidOp());
		}
	}
	
	void THCudaTensor_dsigmoid(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDSigmoidOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDSigmoidOp());
		}
	}
}

#endif
