#include "hip/hip_runtime.h"
#ifndef THTENSOR_CUDA_OPS_H
#define THTENSOR_CUDA_OPS_H

// define some additional CUDA operations
extern "C" {
#include "THCudaTensorOps.h"
}
#include "THC/THCApply.cuh"
#include "THC/THCTensorMath.h"
#include "THC/THCBlas.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#define SOFTMAX_THREADS 128
// Use 1024 threads per block, which requires cuda sm_2x or above
#define UNFOLD_THREADS 1024

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + UNFOLD_THREADS - 1) / UNFOLD_THREADS;
}


struct TensorDTanOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = 1.- (*in) * (*in);
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = 1.- (*v) * (*v);
	  }
};

struct TensorSigmoidOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = 1./(1.+ exp(- *in));
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = 1./(1.+ exp(- *v));
	  }
};

struct TensorDSigmoidOp {
	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (1. - *in) * (*in);
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (1. - *v) * (*v);
	  }
};

struct TensorThresholdOp {
	  TensorThresholdOp(float t, float c, float o) : thresh(t),coeff(c),offset(o) {}

	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (*in) > thresh ? (*in) : coeff * (*in) + offset;
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (*v) > thresh ? (*v) : coeff * (*v) + offset;
	  }
	  
	  const float thresh;
	  const float coeff;
	  const float offset;
};

struct TensorDThresholdOp {
	  TensorDThresholdOp(float t, float c) : thresh(t),coeff(c) {}

	  __device__ __forceinline__ void operator()(float* out, float* in) {
	    *out = (*in) > thresh ? 1 : coeff;
	  }

	  __device__ __forceinline__ void operator()(float* v) {
	    *v = (*v) > thresh ? 1 : coeff;
	  }
	  
	  const float thresh;
	  const float coeff;
};

__global__ void maxpool(float *input, float *output,
                        int input_n, int input_h, int input_w,
                        int kH, int kW, int dH, int dW)
{
	// iterators
	int xx, yy;

	// output size
	const int output_w = (input_w - kW) / dW + 1;
	const int output_h = (input_h - kH) / dH + 1;

	// compute offsets based on thread/block ID
	int o = blockIdx.x;
	int i = o;

	int xx_start = threadIdx.x;
	int xx_end = output_w;
	const int xx_step = blockDim.x;

	int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
	int yy_end = output_h;
	const int yy_step = blockDim.y*gridDim.y;

	// select input/output plane
	output = output + o*output_w*output_h;
	input = input + i*input_w*input_h;

	// For all output pixels...
	for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    	for(xx = xx_start; xx < xx_end; xx+=xx_step) {
			float *ptr_input = input + yy*dH*input_w + xx*dW;
      		float *ptr_output = output + yy*output_w + xx;
      		float max = -FLT_MAX;
      		int kx, ky;
      		for(ky = 0; ky < kH; ky++) {
        		for(kx = 0; kx < kW; kx++) {
          			float val = ptr_input[kx];
          			if (val > max) {
            			max = val;
          			}
        		}
        		ptr_input += input_w; // next input line
      		}
      		// Update output
      		*ptr_output = max;
    	}
  	}
}


// softmax kernel with 128 threads based on cunn Softmax.cu
__global__ void softmax(float *output, float *input, int nframe, int dim)
{
  __shared__ float buffer[ SOFTMAX_THREADS + 1];
  int k = blockIdx.x;
  float *input_k = input + k*dim;
  float *output_k = output + k*dim;

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  // max?
  buffer[threadIdx.x] = -FLT_MAX;
  for (int i=i_start; i<i_end; i+=i_step)
  {
    float z = input_k[i];
    if(buffer[threadIdx.x] < z)
      buffer[threadIdx.x] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float max_k = -FLT_MAX;
    for (int i=0; i<blockDim.x; i++)
    {
      if(max_k < buffer[i])
        max_k = buffer[i];
    }
    buffer[SOFTMAX_THREADS] = max_k;
  }

  __syncthreads();

  // sum?
  float max_k = buffer[SOFTMAX_THREADS];
  buffer[threadIdx.x] = 0;
  for (int i=i_start; i<i_end; i+=i_step) {
    float z = __expf(input_k[i]-max_k);
    buffer[threadIdx.x] += z;
    output_k[i] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    float sum_k = 0;
    for (int i=0; i<blockDim.x; i++)
      sum_k += buffer[i];
    buffer[SOFTMAX_THREADS] = sum_k;
  }

  __syncthreads();

  // softmax
  float sum_k = buffer[SOFTMAX_THREADS];
  for (int i=i_start; i<i_end; i+=i_step)
    output_k[i] = output_k[i] / sum_k;
}

// Kernel for fast unfold+copy
// (borrowed from Caffe: https://github.com/BVLC/caffe/blob/master/src/caffe/layers/conv_layer.cu)
__global__ void im2col_kernel(const int n, const float* data_im,
    const int height, const int width, const int ksize_h, const int ksize_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w, const int height_col, const int width_col,
    float* data_col) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
      index < n;                                      
      index += blockDim.x * gridDim.x){
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize_h * ksize_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize_h; ++i) {
      for (int j = 0; j < ksize_w; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
          data_im[i * width + j] : 0;
          
        data_col += height_col * width_col;
      }
    }
  }
}


extern "C" {
	void THCudaTensor_dtanh(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDTanOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDTanOp());
		}
	}
	
	void THCudaTensor_sigmoid(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorSigmoidOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorSigmoidOp());
		}
	}
	
	void THCudaTensor_dsigmoid(THCState *state, THCudaTensor *dest, THCudaTensor *src)
	{
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDSigmoidOp());
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDSigmoidOp());
		}
	}
	
	void THCudaTensor_threshold(THCState *state, THCudaTensor *dest, THCudaTensor* src, float thresh, float coeff, float of){
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorThresholdOp(thresh, coeff, of));
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorThresholdOp(thresh, coeff, of));
		}
	}

	void THCudaTensor_dthreshold(THCState *state, THCudaTensor *dest, THCudaTensor* src, float thresh, float coeff){
		if (dest == src) {
			THCudaTensor_pointwiseApply1(state, dest, TensorDThresholdOp(thresh, coeff));
		} else {
			THCudaTensor_pointwiseApply2(state, dest, src, TensorDThresholdOp(thresh, coeff));
		}
	}
	
	int THCudaTensor_argmax(THCState *state, THCudaTensor *t){
		t = THCudaTensor_newContiguous(state, t);
		thrust::device_ptr<float> data(THCudaTensor_data(state, t));

		thrust::device_vector<float>::iterator iter =
			thrust::max_element(data, data + THCudaTensor_nElement(state, t));

		int position = thrust::device_pointer_cast(&(iter[0])) - data;
		THCudaTensor_free(state, t);

		return position;
	}

	int THCudaTensor_argmin(THCState *state, THCudaTensor *t){
		t = THCudaTensor_newContiguous(state, t);
		thrust::device_ptr<float> data(THCudaTensor_data(state, t));

		thrust::device_vector<float>::iterator iter =
			thrust::min_element(data, data + THCudaTensor_nElement(state, t));

		int position = thrust::device_pointer_cast(&(iter[0])) - data;
		THCudaTensor_free(state, t);

		return position;
	}
	
	void THCudaTensor_spatialmaxpool(THCState *state, THCudaTensor *output, THCudaTensor *input,
			int kW, int kH, int dW, int dH){	
		
		long nInputCols = input->size[2];
    	long nInputRows = input->size[1];
    	long nInputPlane = input->size[0];
    	long nOutputCols = (nInputCols - kW) / dW + 1;
    	long nOutputRows = (nInputRows - kH) / dH + 1;

    	input = THCudaTensor_newContiguous(state, input);
    	float* input_data = THCudaTensor_data(state, input);

    	THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
    	float* output_data = THCudaTensor_data(state, output);

    	// cuda blocks & threads:
    	int yblocks = (int)(16L / nInputPlane);
    	yblocks = yblocks < 1 ? 1 : yblocks;
    	dim3 blocks(nInputPlane,yblocks);
    	dim3 threads(32,8);

    	// run maxpool kernel
    	maxpool <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (
    	  input_data, output_data,
    	  nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
	
		THCudaTensor_free(state, input);
	}
	
	
	void THCudaTensor_softmax(THCState *state, THCudaTensor *output, THCudaTensor *input){	
    	input = THCudaTensor_newContiguous(state, input);
    	float* input_data = THCudaTensor_data(state, input);
    	float* output_data = THCudaTensor_data(state, output);

    	// cuda blocks & threads:
    	dim3 blocks(1);
    	dim3 threads(SOFTMAX_THREADS);

    	// run softmax kernel
    	softmax<<<blocks,threads,0, THCState_getCurrentStream(state)>>>(
    		output_data, input_data, 1, input->storage->size);
    	
		THCudaTensor_free(state, input);
	}
	
	
	// helper function for unfolding matrix for convolution
	void im2col(THCState *state, const float* data_im, const int channels,
	    const int height, const int width, const int ksize_h, const int ksize_w, const int pad_h,
	    const int pad_w, const int stride_h, const int stride_w, float* data_col) {
	  // We are going to launch channels * height_col * width_col kernels, each
	  // kernel responsible for copying a single-channel grid.
	  int height_col = (height + 2 * pad_h - ksize_h) / stride_h + 1;
	  int width_col = (width + 2 * pad_w - ksize_w) / stride_w + 1;
	  int num_kernels = channels * height_col * width_col;
	  
	  // Launch
	  im2col_kernel <<<GET_BLOCKS(num_kernels), UNFOLD_THREADS, 0, THCState_getCurrentStream(state)>>> (
	      num_kernels, data_im, height, width, ksize_h, ksize_w,
	      pad_h, pad_w, stride_h, stride_w,
	      height_col, width_col, data_col
	  );
	}
	
	void THCudaTensor_spatialconvolve(THCState *state, THCudaTensor *output, THCudaTensor *input,
		THCudaTensor* weight, THCudaTensor* bias, int dW, int dH, int pW, int pH)
	{
		long nOutputPlane = weight->size[0];
		long kW = weight->size[3];
		long kH = weight->size[2];
		long inputWidth   = input->size[2];
  		long inputHeight  = input->size[1];
  		long nInputPlane = input->size[0];
  		long outputWidth  = (inputWidth + 2*pW - kW) / dW + 1;
  		long outputHeight = (inputHeight + 2*pH - kH) / dH + 1;
		
		// create temp tensors for unfolding
		THCudaTensor* columns = THCudaTensor_newWithSize2d(state, nInputPlane*kW*kH, outputHeight*outputWidth);
		THCudaTensor* ones = THCudaTensor_newWithSize2d(state, outputHeight, outputWidth);
		THCudaTensor_fill(state, ones, 1);
		
		
		// Do Bias first:
	    // M,N,K are dims of matrix A and B
	    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
	    long m_ = nOutputPlane;
	    long n_ = outputHeight * outputWidth;
	    long k_ = 1;
	
	    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
	    THCudaBlas_gemm(
	        state,
	        't', 'n',
	        n_, m_, k_,
	        1,
	        THCudaTensor_data(state, ones), k_,
	        THCudaTensor_data(state, bias), k_,
	        0,
	        THCudaTensor_data(state, output), n_
	    );
	
	    // Extract columns:
	    im2col(
	      state,
	      THCudaTensor_data(state, input),
	      nInputPlane, inputHeight, inputWidth, kH, kW, pH, pW, dH, dW,
	      THCudaTensor_data(state, columns)
	    );
	
	    // M,N,K are dims of matrix A and B
	    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
	    long m = weight->size[0];
	    long n = columns->size[1];
	    long k = columns->size[0];
	
	    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
	    THCudaBlas_gemm(
	        state,
	        'n', 'n',
	        n, m, k,
	        1,
	        THCudaTensor_data(state, columns), n,
	        THCudaTensor_data(state, weight), k,
	        1,
	        THCudaTensor_data(state, output), n
	    );
	  

		// free temp tensors
		THCudaTensor_free(state, columns);
		THCudaTensor_free(state, ones);
		
	}
}
#endif
